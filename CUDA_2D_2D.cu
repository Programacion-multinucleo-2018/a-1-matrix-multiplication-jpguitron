#include "hip/hip_runtime.h"
#include "common.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <chrono>

using namespace std;

#define N 1000

//Fill the matrix with natural numbers starting with 0 (row major order)
void fillMatrix(long * matrix)
{
  int i;
  int size = N*N;
  for(i = 0; i < size; i++)
  {
      matrix[i] = i;
  }
}

//Print the matrix
void printMatrix(long * m_r)
{
  int size = N*N;
  int x;
  for(x = 0; x < size; x++)
  {

      if(x%N==0)
      {
        printf("\n");
      }
      printf("%ld ", m_r[x]);
  }
}
//Compare if two matrices are the same
bool checkResult(long * m_host, long * m_gpu)
{
  int size = N*N;
  for(int x = 0; x<size;x++)
  {
    if(m_host[x]!=m_gpu[x])
      return false;
  }
  return true;
}
//multiplication of matrices in cpu
void mulMatrix(long * m_r, long * m1, long * m2)
{
  int x;
  int y;
  int z;
  for(y=0;y<N;y++)
  {
    for(z = 0; z < N; z++)
    {
      for(x = 0; x < N; x++)
      {
          m_r[y*N+z] += m1[x+y*N] * m2[x*N+z];
      }
    }
  }
}
//multiplication of matrices in gpu
__global__ void mulMatrixGPU2D(long *MatA, long *MatB, long *MatC)
{
  unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
  long sum = 0;

  if (ix < N && iy < N)
  {
    for(int in =0;in<N;in++)
    {
        sum += MatA[ix*N+in] * MatB[in*N+iy];
    }
    MatC[ix*N+iy]=sum;
  }
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    SAFE_CALL(hipSetDevice(dev), "Error setting device");

    // set up data size of matrix
    int nx = N;
    int ny = N;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(long);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    long *h_m1, *h_m2, *hostRef, *gpuRef;
    h_m1 = (long *)malloc(nBytes);
    h_m2 = (long *)malloc(nBytes);
    hostRef = (long *)malloc(nBytes);
    gpuRef = (long *)malloc(nBytes);

    // initialize data at host side

    fillMatrix(h_m1);
    fillMatrix(h_m2);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result SAFE_CALLs
    auto start_cpu =  chrono::high_resolution_clock::now();
    mulMatrix(hostRef, h_m1, h_m2);
    auto end_cpu =  chrono::high_resolution_clock::now();
    chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;

    printf("sumMatrixOnHost elapsed %f ms\n", duration_ms.count());

    // malloc device global memory
    long *d_MatA, *d_MatB, *d_MatC;
    SAFE_CALL(hipMalloc((void **)&d_MatA, nBytes), "Error allocating d_MatA");
    SAFE_CALL(hipMalloc((void **)&d_MatB, nBytes), "Error allocating d_MatB");
    SAFE_CALL(hipMalloc((void **)&d_MatC, nBytes), "Error allocating d_MatC");

    // transfer data from host to device
    SAFE_CALL(hipMemcpy(d_MatA, h_m1, nBytes, hipMemcpyHostToDevice), "Error copying d_MatA");
    SAFE_CALL(hipMemcpy(d_MatB, h_m2, nBytes, hipMemcpyHostToDevice), "Error copying d_MatB");

    // invoke kernel at host side
    int dimx = 2;
    int dimy = 256;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    start_cpu =  chrono::high_resolution_clock::now();
    mulMatrixGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC);
    SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
    end_cpu =  chrono::high_resolution_clock::now();
    duration_ms = end_cpu - start_cpu;


    printf("sumMatrixOnGPU2D <<<(%d,%d), (%d,%d)>>> elapsed %f ms\n", grid.x,
           grid.y,
           block.x, block.y, duration_ms.count());

    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");

    // Compare results
    if(checkResult(hostRef, gpuRef))
      printf("They are equal\n");
    else
      printf("They are different\n");

    // free device global memory
    SAFE_CALL(hipFree(d_MatA), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatB), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatC), "Error freeing memory");


    // free host memory
    free(h_m1);
    free(h_m2);
    free(hostRef);
    free(gpuRef);

    // reset device
    SAFE_CALL(hipDeviceReset(), "Error reseting");

    return (0);
}
